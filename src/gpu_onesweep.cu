#include "hip/hip_runtime.h"
#include <iostream>
#include <hipcub/hipcub.hpp>
#include "hip/hip_runtime.h"
#include "init_data.hpp"
#include <vector>
#include <algorithm>
#include "onesweep.hpp"
#include "utils.hpp"
#include "rs_scan.h"
#include "rs_sort.h"
// #define DEBUG

int main(int argc, char* argv[])
{
    if (argc < 2) {
        std::cout << "Usage: program_name count" << std::endl;
        return 1;
    }

    const uint count = 1 << std::stoi(argv[1]);
    std::cout << "Problem size: " << count << std::endl;
    std::vector<uint> h_input(count);
    std::vector<uint> h_output(count);
    initKeys(h_input.data(), count, 0, count);
    std::clock_t start;

// record cpu kernel
    start = std::clock();
    std::sort(h_input.begin(), h_input.end());
    double cpu_duration = (std::clock() - start) / (double)CLOCKS_PER_SEC;
    std::cout << "CPU time: " << cpu_duration << " s" << std::endl;

// record gpu reduce and scan kernel
    uint *d_in, *d_out;
    initKeys(h_input.data(), count, 0, count);
    start = std::clock();
    hipMalloc(&d_in, sizeof(uint) * count);
    hipMalloc(&d_out, sizeof(uint) * count);
    hipMemcpy(d_in, h_input.data(), count * sizeof(uint), hipMemcpyHostToDevice);
    radix_sort(d_out, d_in, count);
    double rs_duration = (std::clock() - start) / (double)CLOCKS_PER_SEC;
    std::cout << "Reduce and Scan time: " << rs_duration << " s" << std::endl;
    hipMemcpy(h_output.data(), d_out, sizeof(uint) * count, hipMemcpyDeviceToHost);
    hipFree(d_out);
    hipFree(d_in);

// record gpu oners_durationsweep kernel
    initKeys(h_input.data(), count, 0, count);
    start = std::clock();
    onesweepRadixSort(h_input.data(), h_output.data(), count);
    double onesweep_time = (std::clock() - start) / (double)CLOCKS_PER_SEC;
    std::cout << "Onesweep: time: " << onesweep_time << " s" << std::endl;

// record cub sort kernel
    // uint *d_in, *d_out;
    initKeys(h_input.data(), count, 0, count);
    start = std::clock();
    hipMalloc((void**)&d_in, count * sizeof(uint));
    hipMalloc((void**)&d_out, count * sizeof(uint));
    hipMemcpy(d_in, h_input.data(), count * sizeof(uint), hipMemcpyHostToDevice);

    // Determine temporary device storage requirements
    void *d_temp_storage = NULL;
    size_t temp_storage_bytes = 0;
    hipcub::DeviceRadixSort::SortKeys(d_temp_storage, temp_storage_bytes, d_in, d_out, count);

    hipMalloc(&d_temp_storage, temp_storage_bytes);

    
    hipcub::DeviceRadixSort::SortKeys(d_temp_storage, temp_storage_bytes, d_in, d_out, count);
    double cub_time = (std::clock() - start) / (double)CLOCKS_PER_SEC;
    std::cout << "CUB time: " << cub_time << " s" << std::endl;
    hipFree(d_out);
    hipFree(d_in);


#ifdef DEBUG
    printVec("Input", h_input);
    printVec("Sorted Output", h_output);
#endif

    return 0;
}
